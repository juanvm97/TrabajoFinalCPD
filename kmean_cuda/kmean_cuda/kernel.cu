#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>

typedef struct
{
	double _r;
	double _g;
	double _b;
	double _m;
	double _n;

} Point;

#define CUDA_CALL(x) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}} 

void readImageSize(FILE* ifp, int* K, int* a, int* b)
{
	fscanf(ifp, "%d\n", K);
	printf("%d\n", *K);

	fscanf(ifp, "%d\n", a);
	printf("%d\n", *a);

	fscanf(ifp, "%d\n", b);
	printf("%d\n", *b);
}

//reads the ifp file and stores in structure
void readPoints(FILE* ifp, Point* points, int num_points)
{
	int i;
	for (i = 0; i < num_points; i++)
	{
		fscanf(ifp, "%lf,%lf,%lf,%lf,%lf", &points[i]._r, &points[i]._g, &points[i]._b, &points[i]._m, &points[i]._n);
		//printf("%lf,%lf,%lf,%lf,%lf\n", points[i]._r, points[i]._g, points[i]._b, points[i]._m, points[i]._n);
	}
}

//Initialize random points as assumed means
void initialize(Point* mean, int K, int num_points, Point* points)
{
	int i, a, p = 2;
	srand(time(NULL));
	for (i = 0; i < K; i++)
	{
		a = num_points / p;
		mean[i]._r = points[a]._r;
		mean[i]._g = points[a]._g;
		mean[i]._b = points[a]._b;
		mean[i]._m = points[a]._m;
		mean[i]._n = points[a]._n;
		p++;
	}
}

//All points having no clusters
void IntClusterMem(int* cluster, int num_points)
{
	int i;
	for (i = 0; i < num_points; i++)
	{
		cluster[i] = -1;
	}
}

//to calculate which cluster is the point belonging to.
__global__ void pointsCluster(int* after_cluster_d, Point* point_d, Point* Dmean, int K, int x, int y)
{
	//__shared__ Point Dmean[105];
	//printf("\n%d\t%d\t%d\n",K,x,y);
	int j, k, i;
	j = blockIdx.x * blockDim.x + threadIdx.x;
	k = blockIdx.y * blockDim.y + threadIdx.y;

	if (j < x && k < y)
	{
		int parent = 0;
		double dist = 0;
		int t = (k * (x)+j);
		//if(t>204790)
		//printf("t = %d\n",t);
		double minDist = sqrt((pow((point_d[t]._r - Dmean[0]._r), 2) + pow((point_d[t]._g - Dmean[0]._g), 2) + pow((point_d[t]._b - Dmean[0]._b), 2) + pow((point_d[t]._m - Dmean[0]._m), 2) + pow((point_d[t]._n - Dmean[0]._n), 2)));
		for (i = 1; i < K; i++)
		{
			dist = sqrt((pow((point_d[t]._r - Dmean[i]._r), 2) + pow((point_d[t]._g - Dmean[i]._g), 2) + pow((point_d[t]._b - Dmean[i]._b), 2) + pow((point_d[t]._m - Dmean[i]._m), 2) + pow((point_d[t]._n - Dmean[i]._n), 2)));
			if (minDist >= dist)
			{
				parent = i;
				minDist = dist;

			}
		}
		after_cluster_d[t] = parent;
	}
}


//calculate new mean
void calcNewMean(Point* points, int* cluster, Point* mean, int K, int num_points)
{
	Point* newMean = (Point*)malloc(sizeof(Point) * K);
	int* members = (int*)malloc(sizeof(int) * (K));
	int i;
	for (i = 0; i < K; i++)
	{
		members[i] = 0;
		newMean[i]._r = 0;
		newMean[i]._g = 0;
		newMean[i]._b = 0;
		newMean[i]._m = 0;
		newMean[i]._n = 0;
	}
	for (i = 0; i < num_points; i++)
	{
		members[cluster[i]]++;
		newMean[cluster[i]]._r += points[i]._r;
		newMean[cluster[i]]._g += points[i]._g;
		newMean[cluster[i]]._b += points[i]._b;
		newMean[cluster[i]]._m += points[i]._m;
		newMean[cluster[i]]._n += points[i]._n;
	}
	for (i = 0; i < K; i++)
	{
		if (members[i] != 0.0)
		{
			newMean[i]._r /= members[i];
			newMean[i]._g /= members[i];
			newMean[i]._b /= members[i];
			newMean[i]._m /= members[i];
			newMean[i]._n /= members[i];
		}
		else
		{
			newMean[i]._r = 0;
			newMean[i]._g = 0;
			newMean[i]._b = 0;
			newMean[i]._m = 0;
			newMean[i]._n = 0;
		}
	}
	for (i = 0; i < K; i++)
	{
		mean[i]._r = newMean[i]._r;
		mean[i]._g = newMean[i]._g;
		mean[i]._b = newMean[i]._b;
		mean[i]._m = newMean[i]._m;
		mean[i]._n = newMean[i]._n;
	}
}

//check for convergence
int chkConvrg(int* before_clusters, int* after_cluster, int num_points, float tol)
{
	int i;
	tol = num_points * tol;
	for (i = 0; i < num_points; i++)
	{
		if (abs(before_clusters[i] - after_cluster[i]) > tol)
		{
			return -1;
		}
	}
	return 0;
}

void createImage2(int* after_cluster, int K, int num_points, Point* points, int width, int height)
{
	Point* colors = new Point[K];
	srand(time(NULL));
	for (int i = 0; i < K; i++)
	{
		colors[i]._m = 0;
		colors[i]._n = 0;
		colors[i]._r = rand() % 255;
		colors[i]._g = rand() % 255;
		colors[i]._b = rand() % 255;
	}

	std::ofstream img("picture1.ppm");

	img << "P3" << std::endl;
	img << width << " " << height << std::endl;
	img << "255" << std::endl;

	for (int y = 0; y < width; y++) {
		for (int x = 0; x < height; x++) {

			//std::cout << points[x * width + y].centroide << std::endl;
			int r = colors[after_cluster[y * height + x]]._r;
			int g = colors[after_cluster[y * height + x]]._g;
			int b = colors[after_cluster[y * height + x]]._b;

			img << r << " " << g << " " << b << std::endl;
		}
	}
}

int main(int argc, char* argv[])
{
	//cpu variables
	int K;
	int num_points;
	int* before_clusters;
	int i;
	int job_done = 0;
	int x, y, iter = 0, iterations;

	Point* mean;
	Point* points;

	int* after_cluster;
	float tol;

	//gpu variables
	Point* points_d;
	Point* mean_d;
	int* after_cluster_d;
	int* before_cluster_d;

	hipEvent_t startinit, endinit, startmean, endmean, startcal, endcal, startindex, endindex;
	hipEvent_t start1, end1;
	float timeinit, timemean, timecal, timeindex;
	float time1;
	//float totTime = 0;
	tol = 0.0;

	hipEventCreate(&start1);
	hipEventCreate(&end1);
	hipEventRecord(start1, 0);

	//Readinf file
	FILE* ifp;
	fopen_s(&ifp, "input2.txt", "r");
	readImageSize(ifp, &K, &x, &y);
	num_points = x * y;
	int blockX = 32;
	int blockY = 32;
	//allocate CPU memory
	points = (Point*)malloc(sizeof(Point) * num_points);
	readPoints(ifp, points, num_points);
	fclose(ifp);
	//printf("Input Read Successfully \n");
	before_clusters = new int[num_points];
	after_cluster = new int[num_points];
	mean = new Point[K];

	//initializing to default values
	initialize(mean, K, num_points, points);
	IntClusterMem(before_clusters, num_points);
	IntClusterMem(after_cluster, num_points);

	//printf("points = %lf",points[0]._r);
	//allocate gpu memory
	//printf("No problem till Here1\n");
	CUDA_CALL(hipMalloc((void**)&after_cluster_d, sizeof(int) * num_points));
	CUDA_CALL(hipMalloc((void**)&before_cluster_d, sizeof(int) * num_points));
	CUDA_CALL(hipMalloc((void**)&points_d, sizeof(Point) * num_points));
	CUDA_CALL(hipMalloc((void**)&mean_d, sizeof(Point) * K));
	//printf("No problem till Here2\n");

	hipEventCreate(&startinit);
	hipEventCreate(&endinit);
	hipEventRecord(startinit, 0);

	//copy data points to device
	CUDA_CALL(hipMemcpy(points_d, points, sizeof(Point) * num_points, hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(after_cluster_d, after_cluster, sizeof(int) * num_points, hipMemcpyHostToDevice));

	hipEventRecord(endinit, 0);
	hipEventSynchronize(endinit);
	hipEventElapsedTime(&timeinit, startinit, endinit);

	while (1)
	{
		//printf("No problem till Here4\n");
		iter++;
		hipEventCreate(&startmean);
		hipEventCreate(&endmean);
		hipEventRecord(startmean, 0);
		//copy initial centroids to device
		CUDA_CALL(hipMemcpy(mean_d, mean, sizeof(Point) * K, hipMemcpyHostToDevice));
		hipEventRecord(endmean, 0);
		hipEventSynchronize(endmean);
		hipEventElapsedTime(&timemean, startmean, endmean);
		//cuda memory copy
		//CUDA_CALL(hipMemcpy(after_cluster_d, after_cluster, sizeof(int)*num_points, hipMemcpyHostToDevice));
		//CUDA_CALL(hipMemcpy(before_cluster_d, before_clusters, sizeof(int)*num_points, hipMemcpyHostToDevice));
		//CUDA_CALL(hipMemcpy(x_d, &x, sizeof(int), hipMemcpyHostToDevice));
		//CUDA_CALL(hipMemcpy(y_d, &y, sizeof(int), hipMemcpyHostToDevice));
		//CUDA_CALL(hipMemcpy(K_d, &K, sizeof(int), hipMemcpyHostToDevice));
		hipEventCreate(&startcal);
		hipEventCreate(&endcal);
		hipEventRecord(startcal, 0);

		dim3 block(blockX, blockY);
		dim3 grid((x + blockX - 1) / blockX, (y + blockY - 1) / blockY);

		pointsCluster <<< grid, block >>> (after_cluster_d, points_d, mean_d, K, x, y);

		//printf("Time taken by parallel portion: %f\n",time);
		//totTime +=time;
		//printf("No problem till Here5\n");
		hipDeviceSynchronize();
		hipEventRecord(endcal, 0);
		hipEventSynchronize(endcal);
		hipEventElapsedTime(&timecal, startcal, endcal);

		hipEventCreate(&startindex);
		hipEventCreate(&endindex);
		hipEventRecord(startindex, 0);

		CUDA_CALL(hipMemcpy(after_cluster, after_cluster_d, sizeof(int) * num_points, hipMemcpyDeviceToHost));
		
		hipEventRecord(endindex, 0);
		hipEventSynchronize(endindex);
		hipEventElapsedTime(&timeindex, startindex, endindex);
		calcNewMean(points, after_cluster, mean, K, num_points);
		//printf("New Centroids are calculated!\n");

		if (chkConvrg(after_cluster, before_clusters, num_points, tol) == 0)
		{
			printf("K-mean algorithm Converged!\n");
			job_done = 1;
		}
		else
		{
			//printf("Not converged!\n");
			for (i = 0; i < num_points; i++)
			{
				//printf("1 after_cluster[%d]=%d, before_clusters[%d]=%d\n",i,after_cluster[i],i,before_clusters[i]);

				before_clusters[i] = after_cluster[i];

				//printf("after_cluster[%d]=%d, before_clusters[%d]=%d\n",i,after_cluster[i],i,before_clusters[i]);
			}
		}

		if (job_done == 1)
			break;

	}
	
	//Outputting to the ofp file
	//createImage2(after_cluster, K, num_points, points, x, y);

	hipEventRecord(end1, 0);
	hipEventSynchronize(end1);
	hipEventElapsedTime(&time1, start1, end1);
	printf("Time for sending initial data from host to device : %f\t sec\n", timeinit / 1000);
	printf("Time for sending new means from host to device : %f\t sec\n", timemean / 1000);
	printf("Time for calculation : %f\t sec\n", timecal / 1000);
	printf("Time for sending new index from device to host : %f\t sec\n", timeindex / 1000);
	printf("Total Time : %f\t sec\n", time1 / 1000);
	printf("Total Iterations = %d\n", iter);
	CUDA_CALL(hipFree(after_cluster_d));
	CUDA_CALL(hipFree(mean_d));
	CUDA_CALL(hipFree(points_d));
	free(before_clusters);
	free(mean);
	free(points);
	free(after_cluster);

	return 0;
}

